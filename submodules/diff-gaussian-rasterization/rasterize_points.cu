/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
LiteRasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& sh,
	const torch::Tensor& campos,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float scale_modifier,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const int degree,
	const bool prefiltered,
	const bool argmax_depth
) {
	if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
		AT_ERROR("means3D must have dimensions (num_points, 3)");
	}
	
	const int P = means3D.size(0);
	const int H = image_height;
	const int W = image_width;

	auto int_opts = means3D.options().dtype(torch::kInt32);
	auto float_opts = means3D.options().dtype(torch::kFloat32);

	torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
	torch::Tensor out_opacity = torch::full({1, H, W}, 0.0, float_opts);
	torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
	torch::Tensor out_depth = torch::full({1, H, W}, 0.0, float_opts);
	
	torch::Device device(torch::kCUDA);
	torch::TensorOptions options(torch::kByte);
	torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
	torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
	torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
	std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
	std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
	std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
	
	int rendered = 0;
	if(P != 0) {
		int M = 0;
		if(sh.size(0) != 0) {
			M = sh.size(1);
		}

		rendered = CudaRasterizer::Rasterizer::lite_forward(
			geomFunc,
			binningFunc,
			imgFunc,
			P, degree, M,
			background.contiguous().data<float>(),
			W, H,
			means3D.contiguous().data<float>(),
			sh.contiguous().data_ptr<float>(),
			colors.contiguous().data<float>(),
			opacity.contiguous().data<float>(), 
			scales.contiguous().data_ptr<float>(),
			scale_modifier,
			rotations.contiguous().data_ptr<float>(),
			cov3D_precomp.contiguous().data<float>(), 
			viewmatrix.contiguous().data<float>(), 
			projmatrix.contiguous().data<float>(),
			campos.contiguous().data<float>(),
			tan_fovx,
			tan_fovy,
			prefiltered,
			argmax_depth,
			out_color.contiguous().data<float>(),
			out_opacity.contiguous().data<float>(),
			out_depth.contiguous().data<float>(),
			radii.contiguous().data<int>());
  	}
  	return std::make_tuple(
		rendered,
		out_color,
		out_opacity,
		radii,
		out_depth
	);
}


std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor,
	torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,  	// [3, H, W]
	const torch::Tensor& means3D,  		// [P, 3]
    const torch::Tensor& colors,  		// [P, 3]
    const torch::Tensor& opacity,  		// [P, 1]
    const torch::Tensor& normal,  		// [P, 3]
    const torch::Tensor& albedo,  		// [P, 3]
    const torch::Tensor& roughness,  	// [P, 1]
    const torch::Tensor& metallic,  	// [P, 1]
	const torch::Tensor& scales,  		// [P, 3]
	const torch::Tensor& rotations,  	// [P, 4]
	const torch::Tensor& cov3D_precomp,	// [P, 6]
	const torch::Tensor& sh,  			// [P, d2, 3]
	const torch::Tensor& campos,  		// [3]
	const torch::Tensor& viewmatrix,  	// [4, 4]
	const torch::Tensor& projmatrix,  	// [4, 4]
	const float scale_modifier,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const int degree,
	const bool prefiltered,
	const bool argmax_depth,
	const bool inference,
	const bool debug
) {
	if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
		AT_ERROR("means3D must have dimensions (num_points, 3)");
	}
	
	const int P = means3D.size(0);
	const int H = image_height;
	const int W = image_width;

	auto int_opts = means3D.options().dtype(torch::kInt32);
	auto float_opts = means3D.options().dtype(torch::kFloat32);

	torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
	torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
	torch::Tensor out_opacity = torch::full({1, H, W}, 0.0, float_opts);
	torch::Tensor out_depth = torch::full({1, H, W}, 0.0, float_opts);
	torch::Tensor out_normal = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
	torch::Tensor out_albedo = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
	torch::Tensor out_roughness = torch::full({1, H, W}, 0.0, float_opts);
	torch::Tensor out_metallic = torch::full({1, H, W}, 0.0, float_opts);
	
	torch::Device device(torch::kCUDA);
	torch::TensorOptions options(torch::kByte);
	torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
	torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
	torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
	std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
	std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
	std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
	
	int rendered = 0;
	if(P != 0) {
		int M = 0;
		if(sh.size(0) != 0) {
			M = sh.size(1);
		}

		rendered = CudaRasterizer::Rasterizer::forward(
			geomFunc,
			binningFunc,
			imgFunc,
			P, degree, M,
			background.contiguous().data<float>(),
			W, H,
			means3D.contiguous().data<float>(),
			sh.contiguous().data_ptr<float>(),
			colors.contiguous().data<float>(),
			opacity.contiguous().data<float>(), 
			normal.contiguous().data<float>(),
			albedo.contiguous().data<float>(),
			roughness.contiguous().data<float>(),
			metallic.contiguous().data<float>(),
			scales.contiguous().data_ptr<float>(),
			scale_modifier,
			rotations.contiguous().data_ptr<float>(),
			cov3D_precomp.contiguous().data<float>(), 
			viewmatrix.contiguous().data<float>(), 
			projmatrix.contiguous().data<float>(),
			campos.contiguous().data<float>(),
			tan_fovx,
			tan_fovy,
			prefiltered,
			argmax_depth,
			inference,
			out_color.contiguous().data<float>(),
			out_opacity.contiguous().data<float>(),
			out_depth.contiguous().data<float>(),
			out_normal.contiguous().data<float>(),
			out_albedo.contiguous().data<float>(),
			out_roughness.contiguous().data<float>(),
			out_metallic.contiguous().data<float>(),
			radii.contiguous().data<int>(),
			debug);
  	}
  	return std::make_tuple(
		rendered,
		out_color,
		radii,
		geomBuffer,
		binningBuffer,
		imgBuffer,
		out_opacity,
		out_depth,
		out_normal,
		out_albedo,
		out_roughness,
		out_metallic
	);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor,
	torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& normal,
	const torch::Tensor& albedo,
	const torch::Tensor& roughness,
	const torch::Tensor& metallic,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& sh,
	const torch::Tensor& campos,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float scale_modifier,
	const float tan_fovx,
	const float tan_fovy,
	const int degree,
    const torch::Tensor& dL_dout_color,
    const torch::Tensor& dL_dout_opacity,
    const torch::Tensor& dL_dout_normal,
    const torch::Tensor& dL_dout_albedo,
    const torch::Tensor& dL_dout_roughness,
    const torch::Tensor& dL_dout_metallic,
	const torch::Tensor& geomBuffer,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const int R,
	const bool debug
) {
	const int P = means3D.size(0);
	const int H = dL_dout_color.size(1);
	const int W = dL_dout_color.size(2);
	
	int M = 0;
	if(sh.size(0) != 0)
	{	
		M = sh.size(1);
	}

	torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
	torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
	torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
	torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
	torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
	torch::Tensor dL_dnormal = torch::zeros({P, 3}, means3D.options());
	torch::Tensor dL_dalbedo = torch::zeros({P, 3}, means3D.options());
	torch::Tensor dL_droughness = torch::zeros({P, 1}, means3D.options());
	torch::Tensor dL_dmetallic = torch::zeros({P, 1}, means3D.options());
	torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
	torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
	torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
	torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
	
	if(P != 0) {  
		CudaRasterizer::Rasterizer::backward(P, degree, M, R,
			background.contiguous().data<float>(),
			W, H, 
			means3D.contiguous().data<float>(),
			sh.contiguous().data<float>(),
			colors.contiguous().data<float>(),
			normal.contiguous().data<float>(),
			albedo.contiguous().data<float>(),
			roughness.contiguous().data<float>(),
			metallic.contiguous().data<float>(),
			scales.data_ptr<float>(),
			rotations.data_ptr<float>(),
			cov3D_precomp.contiguous().data<float>(),
			viewmatrix.contiguous().data<float>(),
			projmatrix.contiguous().data<float>(),
			campos.contiguous().data<float>(),
			radii.contiguous().data<int>(),
			scale_modifier,
			tan_fovx,
			tan_fovy,
			reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
			reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
			reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
			dL_dout_color.contiguous().data<float>(),
    		dL_dout_opacity.contiguous().data<float>(),
			dL_dout_normal.contiguous().data<float>(),
			dL_dout_albedo.contiguous().data<float>(),
			dL_dout_roughness.contiguous().data<float>(),
			dL_dout_metallic.contiguous().data<float>(),
			dL_dmeans2D.contiguous().data<float>(),
			dL_dconic.contiguous().data<float>(),  
			dL_dopacity.contiguous().data<float>(),
			dL_dnormal.contiguous().data<float>(),
			dL_dalbedo.contiguous().data<float>(),
			dL_droughness.contiguous().data<float>(),
			dL_dmetallic.contiguous().data<float>(),
			dL_dcolors.contiguous().data<float>(),
			dL_dmeans3D.contiguous().data<float>(),
			dL_dcov3D.contiguous().data<float>(),
			dL_dsh.contiguous().data<float>(),
			dL_dscales.contiguous().data<float>(),
			dL_drotations.contiguous().data<float>(),
			debug);
	}

	return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dnormal, dL_dalbedo,
		dL_droughness, dL_dmetallic, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
	torch::Tensor& means3D,
	torch::Tensor& viewmatrix,
	torch::Tensor& projmatrix
) { 
	const int P = means3D.size(0);
	
	torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
	if(P != 0)
	{
		CudaRasterizer::Rasterizer::markVisible(P,
			means3D.contiguous().data<float>(),
			viewmatrix.contiguous().data<float>(),
			projmatrix.contiguous().data<float>(),
			present.contiguous().data<bool>());
	}
	
	return present;
}

torch::Tensor depthToNormal(
	const int width, const int height,
	const float focal_x,
	const float focal_y,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& depthMap
) {	
	torch::Tensor normalMap = torch::full({3, height, width}, 0.0, depthMap.options());

	CudaRasterizer::Rasterizer::depthToNormal(
		width, height, focal_x, focal_y,
		viewmatrix.contiguous().data<float>(),
		depthMap.contiguous().data<float>(),
		normalMap.contiguous().data<float>()
	);
  
	return normalMap;
}